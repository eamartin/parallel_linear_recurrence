#include <inttypes.h>
#include <stdio.h>
#include <time.h>
#include "linear_recurrence.h"

#define gpuErrChk(ans) { gpuAssert2((ans), __FILE__, __LINE__); }
void gpuAssert2(hipError_t code, const char *file, int line) {
  if (code != hipSuccess) {
    fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
  }
}

uint64_t nanotime(void) {
  uint64_t billion = 1000 * 1000 * 1000;
  struct timespec t;
  clock_gettime(CLOCK_REALTIME, &t);
  return t.tv_nsec + billion * t.tv_sec;
}

int main(int argc, char **argv) {
  if (argc != 3) {
    fprintf(stderr, "Must pass n_steps and n_dims as args.\n");
    return 1;
  }

  int n_steps = atoi(argv[1]);
  int n_dims = atoi(argv[2]);
  int n_elements = n_dims * n_steps;
  printf("Running on n_steps=%d n_dims=%d\n", n_steps, n_dims);

  float *decays = (float *) calloc(n_elements, sizeof(float));
  for (int i = 0; i < n_elements; i++) {
    decays[i] = .999;
  }
  float *d_decays;
  gpuErrChk(hipMalloc(&d_decays, n_elements * sizeof(float)));
  gpuErrChk(hipMemcpy(d_decays, decays, n_elements * sizeof(float),
		       hipMemcpyHostToDevice));

  float *impulses = (float *) calloc(n_elements, sizeof(float));
  for (int i = 0; i < n_dims; i++) {
    impulses[i + 0 * n_dims] = 2.0;
  }
  float *d_impulses;
  gpuErrChk(hipMalloc(&d_impulses, n_elements * sizeof(float)));
  gpuErrChk(hipMemcpy(d_impulses, impulses,
		       n_elements * sizeof(float), hipMemcpyHostToDevice));

  float *out = (float *) calloc(n_elements, sizeof(float));
  float *d_out;
  gpuErrChk(hipMalloc(&d_out, n_elements * sizeof(float)));
  gpuErrChk(hipMemset(d_out, 0, n_elements * sizeof(float)));

  gpuErrChk(hipDeviceSynchronize());
  uint64_t plr_start = nanotime();
  compute_linear_recurrence(d_decays, d_impulses, NULL, d_out, n_dims, n_steps);
  gpuErrChk(hipDeviceSynchronize());
  uint64_t plr_ns = nanotime() - plr_start;
  printf("PLR: %lu ns\n", plr_ns);

  gpuErrChk(hipDeviceSynchronize());
  uint64_t slr_start = nanotime();
  compute_serial_linear_recurrence(d_decays, d_impulses, NULL, d_out, n_dims, n_steps);
  gpuErrChk(hipDeviceSynchronize());
  uint64_t slr_ns = nanotime() - slr_start;
  printf("PLR: %lu ns\n", plr_ns);

  gpuErrChk(hipMemcpy(out, d_out, n_elements * sizeof(float),
		       hipMemcpyDeviceToHost));

  gpuErrChk(hipFree(d_decays));
  gpuErrChk(hipFree(d_impulses));
  gpuErrChk(hipFree(d_out));
}
